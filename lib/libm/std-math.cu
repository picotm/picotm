#include "hip/hip_runtime.h"
/* This Source Code Form is subject to the terms of the Mozilla Public
 * License, v. 2.0. If a copy of the MPL was not distributed with this
 * file, You can obtain one at http://mozilla.org/MPL/2.0/. */

ceuta_hdrl(#ifndef TANGER_STM_STD_MATH_H);
ceuta_hdrl(#define TANGER_STM_STD_MATH_H);
ceuta_hdrl(#include <math.h>);

#include <math.h>

/* Trigonometric functions
 */

ceuta_pure(double,      sin,  sin,  double x);
ceuta_pure(double,      cos,  cos,  double x);
ceuta_pure(double,      tan,  tan,  double x);
ceuta_pure(float,       sinf, sinf, float x);
ceuta_pure(float,       cosf, cosf, float x);
ceuta_pure(float,       tanf, tanf, float x);
ceuta_pure(long double, sinl, sinl, long double x);
ceuta_pure(long double, cosl, cosl, long double x);
ceuta_pure(long double, tanl, tanl, long double x);

/* Inverse trigonometric functions
 */

ceuta_pure(double,      asin,   asin,   double x);
ceuta_pure(double,      acos,   acos,   double x);
ceuta_pure(double,      atan,   atan,   double x);
ceuta_pure(float,       asinf,  asinf,  float x);
ceuta_pure(float,       acosf,  acosf,  float x);
ceuta_pure(float,       atanf,  atanf,  float x);
ceuta_pure(long double, asinl,  asinl,  long double x);
ceuta_pure(long double, acosl,  acisl,  long double x);
ceuta_pure(long double, atanl,  atanl,  long double x);
ceuta_pure(double,      atan2,  atan2,  double y, double x);
ceuta_pure(float,       atan2f, atan2f, float y, float x);
ceuta_pure(long double, atan2l, atan2l, long double y, long double x);

/* Exponents and logarithms
 */

ceuta_pure(double,      exp,     exp,     double x);
ceuta_pure(double,      exp2,    exp2,    double x);
ceuta_pure(float,       expf,    expf,    float x);
ceuta_pure(float,       exp2f,   exp2f,   float x);
ceuta_pure(long double, expl,    expl,    long double x);
ceuta_pure(long double, exp2l,   exp2l,   long double x);
ceuta_pure(double,      log,     log,     double x);
ceuta_pure(double,      log2,    log2,    double x);
ceuta_pure(double,      log10,   log10,   double x);
ceuta_pure(float,       logf,    logf,    float x);
ceuta_pure(float,       log2f,   log2f,   float x);
ceuta_pure(float,       log10f,  log10f,  float x);
ceuta_pure(long double, logl,    logl,    long double x);
ceuta_pure(long double, log2l,   log2l,   long double x);
ceuta_pure(long double, log10l,  log10l,  long double x);
ceuta_pure(double,      logb,    logb,    double x);
ceuta_pure(float,       logbf,   logbf,   float x);
ceuta_pure(long double, logbl,   logbl,   long double x);
ceuta_pure(double,      ilogb,   ilogb,   double x);
ceuta_pure(float,       ilogbf,  ilogbf,  float x);
ceuta_pure(long double, ilogbl,  ilogbl,  long double x);
ceuta_pure(double,      pow,     pow,     double x, double y);
ceuta_pure(float,       powf,    powf,    float x, float y);
ceuta_pure(long double, powl,    powl,    long double x, long double y);
ceuta_pure(double,      sqrt,    sqrt,    double x);
ceuta_pure(float,       sqrtf,   sqrtf,   float x);
ceuta_pure(long double, sqrtl,   sqrtl,   long double x);
ceuta_pure(double,      cbrt,    cbrt,    double x);
ceuta_pure(float,       cbrtf,   cbrtf,   float x);
ceuta_pure(long double, cbrtl,   cbrtl,   long double x);
ceuta_pure(double,      hypot,   hypot,   double x, double y);
ceuta_pure(float,       hypotf,  hoypotf, float x, float y);
ceuta_pure(long double, hypotl,  hypotl,  long double x, long double y);
ceuta_pure(double,      expm1,   expm1,   double x);
ceuta_pure(float,       expm1fm, expm1fm, float x);
ceuta_pure(long double, expm1l,  expm1l,  long double x);
ceuta_pure(double,      log1p,   log1p,   double x);
ceuta_pure(float,       log1pf,  log1pf,  float x);
ceuta_pure(long double, log1pl,  log1pl,  long double x);

/* Hyperbolic functions
 */

ceuta_pure(double,      sinh,   sinh,   double x);
ceuta_pure(float,       sinhf,  sinhf,  float x);
ceuta_pure(long double, sinhl,  sinhl,  long double x);
ceuta_pure(double,      cosh,   cosh,   double x);
ceuta_pure(float,       coshf,  coshf,  float x);
ceuta_pure(long double, coshl,  coshl,  long double x);
ceuta_pure(double,      tanh,   tanh,   double x);
ceuta_pure(float,       tanhf,  tanhf,  float x);
ceuta_pure(long double, tanhl,  tanhl,  long double x);
ceuta_pure(double,      asinh,  asinh,  double x);
ceuta_pure(float,       asinhf, asinhf, float x);
ceuta_pure(long double, asinhl, asinhl, long double x);
ceuta_pure(double,      acosh,  acosh,  double x);
ceuta_pure(float,       acoshf, acoshf, float x);
ceuta_pure(long double, acoshl, acoshl, long double x);
ceuta_pure(double,      atanh,  atanh,  double x);
ceuta_pure(float,       atanhf, atanhf, float x);
ceuta_pure(long double, atanhl, atanhl, long double x);

/* Special functions
 */

ceuta_pure(double,      erf,     erf,                double x);
ceuta_pure(float,       erff,    erff,               float x);
ceuta_pure(long double, erfl,    erfl,               long double x);
ceuta_pure(double,      erfc,    erfc,               double x);
ceuta_pure(float,       erfcf,   erfcf,              float x);
ceuta_pure(long double, erfcl,   erfcl,              long double x);
ceuta_decl(double,      lgamma,  lgamma,  double x);
ceuta_decl(float,       lgammaf, lgammaf, float x);
ceuta_decl(long double, lgammal, lgammal, long double x);
ceuta_pure(double,      tgamma,  tgamma,             double x);
ceuta_pure(float,       tgammaf, tgammaf,            float x);
ceuta_pure(long double, tgammal, tgammal,            long double x);
ceuta_pure(double,      j0,      j0,                 double x);
ceuta_pure(double,      j1,      j1,                 double x);
ceuta_pure(double,      jn,      jn,                 int n, double x);
ceuta_pure(double,      y0,      y0,                 double x);
ceuta_pure(double,      y1,      y1,                 double x);
ceuta_pure(double,      yn,      yn,                 int n, double x);

double
tanger_stm_std_lgamma(double x)
{
    int sign;

    double res = lgamma_r(x, &sign);

    tanger_stm_tx_t *tx = tanger_stm_get_tx();
    assert(tx);

    tanger_stm_store32(tx, (uint32_t*)&signgam, sign);

    return res;
}

float
tanger_stm_std_lgammaf(float x)
{
    int sign;

    float res = lgamma_r(x, &sign);

    tanger_stm_tx_t *tx = tanger_stm_get_tx();
    assert(tx);

    tanger_stm_store32(tx, (uint32_t*)&signgam, sign);

    return res;
}

long double
tanger_stm_std_lgammal(long double x)
{
    int sign;

    long double res = lgamma_r(x, &sign);

    tanger_stm_tx_t *tx = tanger_stm_get_tx();
    assert(tx);

    tanger_stm_store32(tx, (uint32_t*)&signgam, sign);

    return res;
}

ceuta_hdrl(#endif);

